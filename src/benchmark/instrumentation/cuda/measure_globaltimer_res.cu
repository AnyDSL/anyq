#include <cstdio>
#include <stdexcept>
#include <iostream>

#include <hip/hip_runtime.h>

#include <cupti.h>
#include <cupti_profiler_target.h>
#include <stdexcept>


__device__ unsigned long long timestamp()
{
	unsigned long long timestamp;
	asm volatile("mov.u64 %0, %globaltimer;" : "=l"(timestamp) ::);
	return timestamp;
}

__device__ unsigned long long wait_timestamp(unsigned long long t)
{
	unsigned long long curr_t;

	do
	{
		curr_t = timestamp();
	} while (curr_t == t);

	return curr_t;
}

__global__ void test()
{
	for (int i = 0; i < 128; ++i)
	{
		wait_timestamp(timestamp());

		unsigned long long t_1 = timestamp();
		unsigned long long t_2 = wait_timestamp(t_1);

		printf("%llu ", t_2 - t_1);
	}

	printf("\n");
}

int main()
{
	try
	{
		CUpti_Profiler_Initialize_Params params = {
			.structSize = sizeof(params),
			.pPriv = nullptr
		};

		if (cuptiProfilerInitialize(&params) != CUPTI_SUCCESS)
			throw std::runtime_error("cuptiProfilerInitialize() failed");

		test<<<1,1>>>();

		auto err = hipDeviceSynchronize();

		std::cerr << hipGetErrorString(err) << '\n';
	}
	catch (const std::exception& e)
	{
		std::cerr << "ERROR: " << e.what() << '\n';
		return -1;
	}
	catch (...)
	{
		std::cerr << "ERROR: unknown exception\n";
		return -128;
	}
}
